/*
Author: Yash Saraiya
Class: ECE4122 A
Last Date Modified: November 9th 2023
Description:
Evaluating speed of different memory management methods in CUDA
*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <chrono>
#include <ctime>
#include <hiprand/hiprand_kernel.h>

/*
* function that makes walker take steps
* @Param results: result of distance the walker has travel
* @Param steps: number of steps the walker has to take
* @Param seed: seed for random number generator
*/
__global__ void takeSteps(float* results, int steps, unsigned int seed) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init(seed, tid, 0, &state);
    float x;
    float y;
    for (int i = 0; i < steps; ++i) {
        float dir = hiprand_uniform(&state);
        if (dir < 0.25) {
            ++x;
        } else if (dir <  0.5) {
        	--x;
        } else if (dir <  0.75) {
            ++y;
        } else {
            --y;
        }   
    }
    results[tid] = (x*x)+(y*y);
}

/*
* function that makes walker take steps using normal memory allocation
* @Param numWalkers: number of walkers
* @Param steps: number of steps the walker has to take
* @Param seed: seed for random number generator
* @Param threadsPerBlock: threads Per Block
* @Param blocksPerGrid: blocks Per Grid
* @Param print: if code should print the avg dist and time
*/
void cudaMallocNormal(int steps, int numWalkers, int threadsPerBlock, int blocksPerGrid, bool print) {
    float* d_results;
    float* h_results;
    int bytes = sizeof(float) * numWalkers;

    //timer starts
    auto start = std::chrono::high_resolution_clock::now();

    h_results = (float*)malloc(bytes);
    hipMalloc((void**)&d_results, bytes);
    hipMemcpy(d_results, h_results, bytes, hipMemcpyHostToDevice);
    takeSteps<<<blocksPerGrid, threadsPerBlock>>>(d_results, steps, time(NULL));
    hipMemcpy(h_results, d_results, bytes, hipMemcpyDeviceToHost);    

    // calc avg distance
    float sum = 0;
    for (int i=0; i < numWalkers; ++i) {
        sum += sqrt(h_results[i]);
    }

    hipFree(d_results);
    free(h_results);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end-start;
    // prints
    if(print){
        std::cout << "\tTime to calculate(microsec): " << duration.count()*1000000 <<std::endl;
        std::cout << "\tAverage distance from origin: " << sum/numWalkers << std::endl;
    }
}

/*
* function that makes walker take steps using pinned memory allocation
* @Param numWalkers: number of walkers
* @Param steps: number of steps the walker has to take
* @Param seed: seed for random number generator
* @Param threadsPerBlock: threads Per Block
* @Param blocksPerGrid: blocks Per Grid
* @Param print: if code should print the avg dist and time
*/
void hipHostMalloc(int steps, int numWalkers, int threadsPerBlock, int blocksPerGrid, bool print) {
    float* d_results;
    float* h_results;
    int bytes = sizeof(float) * numWalkers;

    //timer starts
    auto start = std::chrono::high_resolution_clock::now();

    hipHostMalloc((void**)&h_results, bytes, hipHostMallocDefault);
    hipMalloc((void**)&d_results, bytes);
    //cudaMemcpy(d_results, h_results, bytes, cudaMemcpyHostToDevice);
    takeSteps<<<blocksPerGrid, threadsPerBlock>>>(d_results, steps, time(NULL));
    hipMemcpy(h_results, d_results, bytes, hipMemcpyDeviceToHost);  

    // calc avg distance
    float sum = 0;
    for (int i=0; i < numWalkers; ++i) {
        sum += sqrt(h_results[i]);
    }

    // free memory
    hipFree(d_results);
    hipHostFree(h_results);


     // timer stops
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end-start;

    // prints
   if(print){
        std::cout << "\tTime to calculate(microsec): " << duration.count()*1000000 <<std::endl;
        std::cout << "\tAverage distance from origin: " << sum/numWalkers << std::endl;
    }
}

/*
* function that makes walker take steps using managed memory allocation
* @Param numWalkers: number of walkers
* @Param steps: number of steps the walker has to take
* @Param seed: seed for random number generator
* @Param threadsPerBlock: threads Per Block
* @Param blocksPerGrid: blocks Per Grid
* @Param print: if code should print the avg dist and time
*/
void hipMallocManaged(int steps, int numWalkers, int threadsPerBlock, int blocksPerGrid, bool print) {
    float* m_results;
    int bytes = sizeof(float) * numWalkers;

    //timer starts
    auto start = std::chrono::high_resolution_clock::now();
    hipMallocManaged(&m_results, bytes);
    takeSteps<<<blocksPerGrid, threadsPerBlock>>>(m_results, steps, time(NULL));
    hipDeviceSynchronize();;    

    // calc avg distance
    float sum = 0;
    for (int i=0; i < numWalkers; ++i) {
        sum += sqrt(m_results[i]);
    }

    hipFree(m_results);

    // timer stops
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end-start;
    // prints
    if(print){
        std::cout << "\tTime to calculate(microsec): " << duration.count()*1000000 <<std::endl;
        std::cout << "\tAverage distance from origin: " << sum/numWalkers << std::endl;
    }
    
}

// main program that executes and calls other functions
int main(int argc, char **argv) {
    int steps = 10000; //default
    int numWalkers = 1000;
    
    for (int i = 1; i < argc; ++i) {
        if (std::string(argv[i]) == "-I" && i + 1 < argc) {
            steps = std::atoi(argv[i + 1]);
            ++i;
        } else if (std::string(argv[i]) == "-W" && i + 1 < argc) {
            numWalkers = std::atoi(argv[i + 1]);
            ++i;
        } else {
            std::cerr << "Usage: " << argv[0] << " -I <integer> -W <integer>" << std::endl;
            return 1;
        }
    }
    std::cout << "\nLab 4 -W " << numWalkers << " -I " << steps << std::endl;
	int threadsPerBlock = 256;
	int blocksPerGrid = (numWalkers + threadsPerBlock) / threadsPerBlock;
    std::cout << "Normal CUDA memory Allocation: " <<std::endl;
    cudaMallocNormal(steps, numWalkers, threadsPerBlock, blocksPerGrid, false); // warm up
    cudaMallocNormal(steps, numWalkers, threadsPerBlock, blocksPerGrid, true);
    std::cout << "Pinned CUDA memory Allocation: " <<std::endl;
    hipHostMalloc(steps, numWalkers, threadsPerBlock, hipHostMallocDefault, blocksPerGrid, false); // warm up
    hipHostMalloc(steps, numWalkers, threadsPerBlock, hipHostMallocDefault, blocksPerGrid, true);
    std::cout << "Managed CUDA memory Allocation: " <<std::endl;
    hipMallocManaged(steps, numWalkers, threadsPerBlock, blocksPerGrid, false); // warm up
    hipMallocManaged(steps, numWalkers, threadsPerBlock, blocksPerGrid, true);
    return 0;
}